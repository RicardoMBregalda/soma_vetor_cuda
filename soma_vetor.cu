
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10240  // Valor de N maior que o limite de threads por bloco

// Kernel para a soma de vetores
__global__ void somaVetores(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {  // Verifica se o índice está dentro dos limites
        c[i] = a[i] + b[i];
    }
}

int main() {
    int *a, *b, *c;             // Vetores no host
    int *d_a, *d_b, *d_c;       // Vetores no device
    int size = N * sizeof(int);

    // Aloca memória no host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Inicializa os vetores no host
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Aloca memória no device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copia os vetores do host para o device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define o número de threads por bloco e o número de blocos
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Executa o kernel para a soma de vetores
    somaVetores<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copia o resultado do device para o host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Imprime alguns resultados para verificação
    for (int i = 0; i < 10; i++) {
        std::cout << "c[" << i << "] = " << c[i] << std::endl;
    }

    // Libera a memória alocada no device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Libera a memória alocada no host
    free(a);
    free(b);
    free(c);

    return 0;
}
